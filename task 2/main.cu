#include "stdio.h"
#include "hip/hip_runtime.h"
#include "stdlib.h"

const int vertical = 64;
const int horizontal = 8;
const int N = vertical * horizontal;

__global__ void add(int* a, int* b, int* c) {
    int thread_idx = (blockIdx.x + blockIdx.y * gridDim.x)*(blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    if (thread_idx < N) {
        c[thread_idx] = a[thread_idx] + b[thread_idx];
    }
}

int main() {
    int* host_a;
    int* host_b;
    int* host_c;

    int* dev_a;
    int* dev_b;
    int* dev_c;

    // allocate memory on host
    hipHostAlloc((void**)&host_a, N * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, N * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, N * sizeof(int), hipHostMallocDefault);

    // allocate memory on device
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++) {
        host_a[i] = -i + 1;
        host_b[i] = i * i;
    }

    // copy data to device
    hipMemcpy((void*)dev_a, (void*)host_a, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy((void*)dev_b, (void*)host_b, sizeof(int)*N, hipMemcpyHostToDevice);
    
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    int threadX = 8;
    int threadY = 4;
    dim3 blocks(N / threadX, N / threadY);
    dim3 threads(threadX, threadY);
    add<<<blocks,threads>>>(dev_a, dev_b, dev_c);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    printf("GPU compute time: %f\n", time);
    hipEventRecord(stop);

    hipMemcpy((void*)host_c, (void*)dev_c, sizeof(int)*N, hipMemcpyDeviceToHost);

    // display the results
    for (int i=0; i<N; i++) {
        printf("%d ", host_c[i]);
    }

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
