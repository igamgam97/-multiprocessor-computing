#include "stdio.h"
#include "hip/hip_runtime.h"
#include "stdlib.h"

const int N = 1024;

__global__ void addNN(int* a, int* b, int* c) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void add11(int* a, int* b, int* c) {
    for (int i = 0; i < N; i++) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int* host_a;
    int* host_b;
    int* host_c;

    int* dev_a;
    int* dev_b;
    int* dev_c;

    // allocate memory on host
    hipHostAlloc((void**)&host_a, N * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, N * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, N * sizeof(int), hipHostMallocDefault);

    // allocate memory on device
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++) {
        host_a[i] = -i + 1;
        host_b[i] = i * i;
    }

    // copy data to device
    hipMemcpy((void*)dev_a, (void*)host_a, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy((void*)dev_b, (void*)host_b, sizeof(int)*N, hipMemcpyHostToDevice);
    
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    add11<<<1,1>>>(dev_a, dev_b, dev_c);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    printf("GPU compute time: %f\n", time);
    hipEventRecord(stop);

    hipMemcpy((void*)host_c, (void*)dev_c, sizeof(int)*N, hipMemcpyDeviceToHost);

    // display the results
    for (int i=0; i<N; i++) {
        printf("%d ", host_c[i]);
    }

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
